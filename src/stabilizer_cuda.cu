#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <iostream> 

// CUDA kernel for box filter convolution
__global__ void boxFilterKernel(const float* padded, float* result, int r, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        float sum = 0.0f;
        for (int k = 0; k < r; ++k) {
            sum += padded[i + k];
        }
        result[i] = sum / r;
    }
}

std::vector<float> boxFilterConvolveCUDA(const std::vector<float> &path, int r) {
    if (path.empty()) return {};

    // Compute median on host
    std::vector<float> tmp = path;
    std::nth_element(tmp.begin(), tmp.begin() + tmp.size()/2, tmp.end());
    float med = tmp[tmp.size()/2];

    // Create padded array
    std::vector<float> padded(path.size() + 2*r, med);
    for(size_t i = 0; i < path.size(); ++i) {
        padded[r + i] = path[i];
    }

    // Allocate device memory
    float *d_padded = nullptr;
    float *d_result = nullptr;
    size_t padded_size = padded.size() * sizeof(float);
    size_t result_size = path.size() * sizeof(float);

    hipMalloc(&d_padded, padded_size);
    hipMalloc(&d_result, result_size);

    // Copy padded data to device
    hipMemcpy(d_padded, padded.data(), padded_size, hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (path.size() + threadsPerBlock - 1) / threadsPerBlock;
    boxFilterKernel<<<blocksPerGrid, threadsPerBlock>>>(d_padded, d_result, r, path.size());

    // Check for kernel launch errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_padded);
        hipFree(d_result);
        return {};
    }

    // Copy result back to host
    std::vector<float> result(path.size());
    hipMemcpy(result.data(), d_result, result_size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_padded);
    hipFree(d_result);

    return result;
}

